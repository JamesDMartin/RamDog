#include "hip/hip_runtime.h"
// CUDA-C includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include "cuprintf.cu"

/* DEFINITIONS */
/* STRUCTURES */

/* FUNCTION PROTOTYPES */
void initDevice();
__global__ void byCol(double *dev_input_data, double *dev_ft_data, int nRows, int nCols);
__global__ void byRow(double *dev_input_data, double *dev_ft_data, int nRows, int nCols);

/* GLOBAL VARS */

/* FUNCTION IMPLEMENTATION */
/////////////////////////////////////////////////////////////////////////
// get cuda info to set the device as the one with the most cuda cores //
/////////////////////////////////////////////////////////////////////////
void initDevice() {

	int cudaDeviceCount, 
		cudaDeviceToUse, 
		*numCudaCores, 
		maxNumCudaCores;
	
	hipDeviceProp_t deviceProp;
	cudaDeviceCount = hipGetDeviceCount(&cudaDeviceCount);
	cudaDeviceToUse = 0;
	numCudaCores = (int *) malloc(sizeof(int) * cudaDeviceCount);
	maxNumCudaCores = 0;
	for(int i = 0; i < cudaDeviceCount; i++) {
		hipGetDeviceProperties(&deviceProp, i);	// get the device properties with a specific cuda call
		numCudaCores[i] = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount;	// get the number of cuda cores for device "i"
		if(numCudaCores[i] > maxNumCudaCores) {
			cudaDeviceToUse = i;
			maxNumCudaCores = numCudaCores[i];
		}
	}
	hipSetDevice(cudaDeviceToUse);
}

/* DEVICE CALLS */

__global__ void byCol(double *dev_input_data_re, double *dev_input_data_im, 
						double *dev_ft_data_re, double *dev_ft_data_im, 
						int nRows, int nCols) {
	int kx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int ky = (blockIdx.y * blockDim.y) + threadIdx.y;
	int ft_idx = kx + ky * nCols;
	cuPrintf("\nx y ft_idx: %d %d", kx, ky, ft_idx);
	int data_idx = 0;
	double PI = 3.14159265359;
	if(kx < nCols && ky < nRows) {
		for(int t = 0; t < nRows; t++) {
			data_idx = t * nCols + kx;
			double arg = 2*PI*t*ky/((double) nRows);
			dev_ft_data_re[ft_idx] +=    dev_input_data_re[data_idx] * cos(arg) + dev_input_data_im[data_idx] * sin(arg);
			dev_ft_data_im[ft_idx] += -1*dev_input_data_re[data_idx] * sin(arg) + dev_input_data_im[data_idx] * cos(arg);
		}
	}
}

__global__ void byRow(double *dev_input_data_re, double *dev_input_data_im, 
						double *dev_ft_data_re, double *dev_ft_data_im, 
						int nRows, int nCols) {
	int kx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int ky = (blockIdx.y * blockDim.y) + threadIdx.y;
	int ft_idx = kx + ky * nCols;
	cuPrintf("\nx y ft_idx: %d %d", kx, ky, ft_idx);
	int data_idx = 0;
	double PI = 3.14159265359;
	if(kx < nCols && ky < nRows) {
		for(int t = 0; t < nCols; t++) {
			data_idx = ky * nCols + t;
			double arg = 2*PI*t*kx/((double) nCols);
			dev_ft_data_re[ft_idx] +=    dev_input_data_re[data_idx] * cos(arg) + dev_input_data_im[data_idx] * sin(arg);
			dev_ft_data_im[ft_idx] += -1*dev_input_data_re[data_idx] * sin(arg) + dev_input_data_im[data_idx] * cos(arg);
		}
	}
}

void emulateByRow(double *input_data, double *ft_data, int nRows, int nCols, int kx, int ky) {
	int ft_idx = kx + ky * nCols;
	int t;
	int data_idx = 0;
	double PI = 3.14159265359;
	if(kx < nCols && ky < nRows) {
		//printf("\nx y ft_idx: %d %d %d: dataIdx: ", x, y, ft_idx);
		double sumReal = 0, sumImag = 0;
		for(t = 0; t < nCols; t++) {
			data_idx = t + ky * nCols;
			//printf(" %d", data_idx);
			sumReal +=    input_data[data_idx] * cos(((double) 2*PI * t * kx) / ((double) nCols));
			sumImag += -1*input_data[data_idx] * sin(((double) 2*PI * t * kx) / ((double) nCols));
		}
		//printf("\nsumReal %lf sumImag %lf", sumReal, sumImag);
		ft_data[ft_idx] = sqrt(pow(sumReal, 2) + pow(sumImag, 2));
	}
}

void emulate(dim3 numBlocks, dim3 threadsPerBlock, double *data, double *host_ft_data, int nRows, int nCols) {
	int kx, ky, x1, y1, x2, y2;
	for(x1 = 0; x1 < numBlocks.x; x1++) {
		for(y1 = 0; y1 < numBlocks.y; y1++) {
			for(x2 = 0; x2 < threadsPerBlock.x; x2++) {
				for(y2 = 0; y2 < threadsPerBlock.y; y2++) {
					kx = (x1 * threadsPerBlock.x) + x2;
					ky = (y1 * threadsPerBlock.y) + y2;
					emulateByRow(data, host_ft_data, nRows, nCols, kx, ky);
				}
			}
		}
	}
}
void printArray(double *data, int nRows, int nCols) {
	int i, j, idx;
	for(i = 0; i < nRows; i++) {
		for(j = 0; j < nCols; j++) {
			idx = j + i * nCols;
			printf("%lf\t", data[idx]);
		}
		printf("\n");
	}
}
/* HOST CALLS */
#ifdef __cplusplus
extern "C"
{
#endif
void runDFT(double *data, double *host_ft_data_re, double *host_ft_data_im, int nRows, int nCols, int colFirst) {
	dim3 threadsPerBlock(8, 8);
	dim3 numBlocks(nCols / threadsPerBlock.x+1,
					nRows / threadsPerBlock.y+1);
	
	double *dev_input_data_re, *dev_input_data_im, *dev_ft_data_re, *dev_ft_data_im;

	size_t array_size = sizeof(double) * nRows * nCols;
	int idx;
	
	/* ALLOCATE DEVICE MEMORY */
	hipMalloc((void **) &dev_input_data_re, array_size);
	hipMalloc((void **) &dev_input_data_im, array_size);
	hipMalloc((void **) &dev_ft_data_re, array_size);
	hipMalloc((void **) &dev_ft_data_im, array_size);
	
	/* INITIALIZE DEVICE MEMORY */
	hipMemset(dev_ft_data_re, 0, array_size);
	hipMemset(dev_ft_data_im, 0, array_size);
	hipMemset(dev_input_data_im, 0, array_size);
	memset(host_ft_data_re, 0, array_size);
	memset(host_ft_data_im, 0, array_size);
	
	printf("\nthreadsPerBlock (%d, %d), numBlocks (%d, %d)", threadsPerBlock.x, threadsPerBlock.y, numBlocks.x, numBlocks.y);
	printf("\nCommencing Print before copy to device");
	printArray(data, 1, nCols);
	/* COPY DATA TO DEVICE */	
	hipMemcpy(dev_input_data_re, data, array_size, hipMemcpyHostToDevice);
	if(colFirst == 0) {
		byRow<<<numBlocks, threadsPerBlock>>>(dev_input_data_re, dev_input_data_im, dev_ft_data_re, dev_ft_data_im, nRows, nCols);
		hipMemcpy(dev_input_data_re, dev_ft_data_re, array_size, hipMemcpyDeviceToDevice);
		hipMemcpy(dev_input_data_im, dev_ft_data_im, array_size, hipMemcpyDeviceToDevice);
		hipMemset(dev_ft_data_re, 0, array_size);
		hipMemset(dev_ft_data_im, 0, array_size);
		byCol<<<numBlocks, threadsPerBlock>>>(dev_input_data_re, dev_input_data_im, dev_ft_data_re, dev_ft_data_im, nRows, nCols);
	} else if(colFirst == 1){
		byCol<<<numBlocks, threadsPerBlock>>>(dev_input_data_re, dev_input_data_im, dev_ft_data_re, dev_ft_data_im, nRows, nCols);
		hipMemcpy(dev_input_data_re, dev_ft_data_re, array_size, hipMemcpyDeviceToDevice);
		hipMemcpy(dev_input_data_im, dev_ft_data_im, array_size, hipMemcpyDeviceToDevice);
		hipMemset(dev_ft_data_re, 0, array_size);
		hipMemset(dev_ft_data_im, 0, array_size);
		byRow<<<numBlocks, threadsPerBlock>>>(dev_input_data_re, dev_input_data_im, dev_ft_data_re, dev_ft_data_im, nRows, nCols);
	} else {
		hipMemcpy(dev_ft_data_re, dev_input_data_re, array_size, hipMemcpyDeviceToDevice);
	}
	hipDeviceSynchronize();
	/* COPY DATA FROM DEVICE */
	hipMemcpy(host_ft_data_re, dev_ft_data_re, array_size, hipMemcpyDeviceToHost);
	hipMemcpy(host_ft_data_im, dev_ft_data_im, array_size, hipMemcpyDeviceToHost);
	
	printf("\nCommencing Print after data has been copied from device");
	printf("\nre:\n");
	printArray(host_ft_data_re, 1, nCols);
	printf("\nim:\n");
	printArray(host_ft_data_im, 1, nCols);

	/* FREE DEVICE MEMORY */
	hipFree(dev_input_data_re);
	hipFree(dev_input_data_im);
	hipFree(dev_ft_data_re);
	hipFree(dev_ft_data_im);
}
#ifdef __cplusplus
}
#endif